#include "hip/hip_runtime.h"
#include<iostream>
#include<fstream>
#include<string>
#include<hip/hip_runtime.h>
#include<random> 
#include<stack>
#include <malloc.h>
#include <stdlib.h>
#include <stdio.h>
#include <string>
#include <time.h>
#include <hip/hip_runtime.h>
#include <math.h>
using namespace std;
//larger than pow(2,25) less than pow(2,31)
#define Prime  149672669

//unsigned int  hashshift(unsigned int a)
//{
//	srand((unsigned)time(Null));
//	a = (a + rand()) + (a << 12);
//	a = a % (int)pow(2, 24)*1.2;
//	if (a < 0)
//		a += ((int)pow(2, 24)*1.2);
//	return a;
//}
__global__ void hashcal(int* d_input, int* d_hash_fun, int* d_a, int* d_b, int table_size, int set_size,int t)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j;
	if (i < set_size)
	{
		for (j = 0; j < t; j++)
		{
			d_hash_fun[i*t+j] = ((d_a[j] * d_input[i] + d_b[j]) % Prime) % table_size;
			//printf(" %d ", d_hash_fun[i*t + j]);
			
		}
		__syncthreads();
		if (i == set_size - 1)
			printf("qaq");

	}

}

__global__ void hashfun(int* d_input, int* d_hash_table, int* d_hash_fun_i, int* d_a, int* d_b,int table_size,int * d_evict,int set_size,int *d_hash_fun,int t)
{
	//calculate first or calculate them each time?
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int key_fun = d_hash_fun_i[i];
	//printf("(d_a[key_fun] * d_input[i] + d_b[key_fun]) mod Prime modtable_size=%d * %d+ %d) mod %d mod %d\n", d_a[key_fun], d_input[i], d_b[key_fun], Prime, table_size);
	int hash_val;
	//printf("thread_i %d,i %d,key_fun %d,hash_val %d\n", thread_i, i, key_fun, hash_val);
	//printf("%d,%d,%d,%d\n", sizeof(int), sizeof(d_a), sizeof(d_hash_table), hash_val);
	//printf("(d_a[key_fun] * d_input[i] + d_b[key_fun]) mod Prime modtable_size=%d * %d+ %d) mod %d mod %d\n", d_a[key_fun], d_input[i], d_b[key_fun], Prime, table_size);
	if (i < set_size)
	{
		hash_val = d_hash_fun[i*t + key_fun];
		if ((d_hash_table[hash_val] == -1 || d_evict[i] == 1) && d_input[i] != 0) {
			d_hash_table[hash_val] = d_input[i];
		}
		__syncthreads();
		//printf("hashtable[i],%d\n", d_hash_table[i]);
		//printf("1qwq%d", *out_num);
		//printf("Hello thread %d, f=%d\n", i);
	}

}

__global__ void evicter(int * d_hash_table, int * d_hash_fun_i,
	int *d_input, int *d_evict ,int *d_a, int *d_b, int table_size,int t, int set_size, int *d_outnum,int *d_hash_fun)
{
	__syncthreads();
	int thread_i = blockDim.x * blockIdx.x + threadIdx.x;
	int key_fun = d_hash_fun_i[thread_i];
	int hash_val;
	//printf("qwq%d,%d\n", d_hash_table[hash_val], d_input[thread_i]);
	if (thread_i < set_size)
	{
		hash_val = d_hash_fun[thread_i*t + key_fun];
		if ((d_hash_table[hash_val] != d_input[thread_i]) && (d_input[thread_i] != 0))
		{
			//printf("qwqhengheng%d\n", thread_i);
			d_evict[thread_i] = 1;
			d_hash_fun_i[thread_i] = (d_hash_fun_i[thread_i] + 1) % t;
			atomicAdd(d_outnum, 1);
		}
		else
		{
			d_evict[thread_i] = 0;

		}
	}
}

//use the most simple to avoid any error
void checkright(int *hash_table, int * hash_fun_i, int * input, int set_size,int table_size,int *a,int *b)
{
	int errcount = 0;
	int key_fun, hash_val;
	for (int i = 0; i < set_size; i++)
	{
		key_fun = hash_fun_i[i];
		hash_val= (a[key_fun] * input[i] + b[key_fun]) % Prime % table_size;
		//printf("(a[key_fun] * input[i] + b[key_fun]) mod Prime modtable_size=%d * %d+ %d) mod %d mod %d\n", a[key_fun], input[i], b[key_fun], Prime, table_size);
		//cout << "hashval " << hash_val << endl;
		//cout << "hash_table[hash_val] " << hash_table[hash_val] << "input[i] " << input[i]<<endl;
		//if (i > set_size-5)
		//{
		//	cout << "here " <<i<<"hashtab " << hash_table[hash_val] << "inpuit" << input[i] << "hash" << hash_val << endl;
		//	cout << _msize(hash_table) / sizeof(hash_table[0]) << endl;
		//	cout << _msize(input) / sizeof(input[0]) << endl;
		//}
		//if (i == 5) exit(0);
		if (hash_table[hash_val] != input[i])
		{
			errcount++;
		}
	}
	if (errcount == 0)
	{
		cout << "DONEqwq\n";
	}
	else {
		cout << "errcount is \n" << errcount;
	}
	
}

void cudacheck(hipError_t err) {
	if (err != hipSuccess) {
		cout << "Could not copy global_flag to CUDA device\n" << hipGetErrorString(err) << endl;
		exit(0);
	}
}

int main/*qwq*/(/*int table, int set,int *input,int bound_in*/) {

	int table_size = (int)pow(2,24);
	//int table_size = table;
	int set_size = pow(2, 10);
	const int t = 3;
	int a[t], b[t];
	int round = 0;
	int outnum = pow(2, 24);
	int * hash_table = new int[table_size];

	int * input = new int[set_size];

	int * hash_fun_i = new int[set_size];
	//int * hash_fun = new int[set_size*t];
	int * hash_fun = new int[set_size*t];
	int * evict = new int[set_size];
	int sizeof_hashtable = sizeof(int)*table_size;
	int sizeof_const = sizeof(int)* t;
	int sizeof_input = sizeof(int)* set_size;
	int *d_a, *d_b, *d_hash_table, *d_input,*d_hash_fun_i,* d_outnum;
	int *d_evict, *d_hash_fun;
	int bound = 0;
	bool needhash = true;
	
	stack <float>stk;
	hipError_t err = hipSuccess;
	bound = (int)4 * log(table_size);
	//bound = bound_in;
	//use -1 to know it is empty for the random number is larger or equal to 0
	fill(hash_table, hash_table + table_size , -1);
	fill(hash_fun, hash_fun + set_size*t , -1);
	fill(evict, evict + set_size , 0);
	fill(hash_fun_i, hash_fun_i + set_size , 0);
	//Use std random number for rand() is not good and I suppose that the input numbers
	//have no equal number. So std random is better.
	std::default_random_engine random(time(NULL));
	//For a multiple input may be larger than INT32MAX,so let them 
	//less than INT32MAX
	std::uniform_int_distribution<int> ranab(1, pow(2,6));
	std::uniform_int_distribution<int> ranin(1, pow(2,25));
	
	for (int i = 0; i < set_size; i++)
		input[i] = ranin(random);
	for (int i = 0; i < t; i++)
	{
		a[i] = ranab(random);
		b[i] = ranab(random)-1;
		//b can be 0 but a can not.
	}
	//for (int i = 0; i < set_size; i++)
	//{
	//	for (int j = 0; j < t; j++)
	//	{
	//		hash_fun[i*t + j] = hash32shift(input[i]);
	//		//cout << hash_fun[i*t + j];
	//	}
	//}
	hipMalloc((void **)&d_hash_table, sizeof_hashtable);
	hipMalloc((void **)&d_input, sizeof_input);
	hipMalloc((void **)&d_hash_fun_i, sizeof_input);
	hipMalloc((void **)&d_hash_fun, sizeof_input*t);
	hipMalloc((void **)&d_a, sizeof_const);
	hipMalloc((void **)&d_b, sizeof_const);
	hipMalloc((void **)&d_outnum, sizeof(int));
	hipMalloc((void **)&d_evict, sizeof_input);
	err = hipMemcpy(d_hash_table, hash_table,sizeof_hashtable, hipMemcpyHostToDevice);cudacheck(err);
	err = hipMemcpy(d_input, input,sizeof_input, hipMemcpyHostToDevice);cudacheck(err);
	err = hipMemcpy(d_hash_fun_i, hash_fun_i, sizeof_input, hipMemcpyHostToDevice);cudacheck(err);
	err = hipMemcpy(d_a, a, sizeof_const, hipMemcpyHostToDevice);cudacheck(err);
	err = hipMemcpy(d_b, b, sizeof_const, hipMemcpyHostToDevice);cudacheck(err);
	err = hipMemcpy(d_hash_fun, hash_fun, sizeof_input*t, hipMemcpyHostToDevice);	cudacheck(err);
	err = hipMemcpy(d_evict, evict, sizeof_input, hipMemcpyHostToDevice); cudacheck(err);
	
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	int total_round = 0;

	while (outnum!=0 && total_round <=15) {
		outnum = 0;
		err =  hipMemcpy(d_outnum, &outnum, sizeof(int), hipMemcpyHostToDevice); cudacheck(err);
		float elapsedTime = 0;

		if (round == bound) {
			round = 0;
			total_round++;
			for (int i = 0; i < t; i++) {
				a[i] = ranab(random);
				b[i] = ranab(random) - 1;
			}
			needhash = true;
			fill(hash_table, hash_table + table_size , -1);
			fill(evict, evict + set_size , 0);
			fill(hash_fun_i, hash_fun_i + set_size , 0);
			fill(hash_fun, hash_fun + set_size * t, -1);
			
			err = hipMemcpy(d_a, a, sizeof_const, hipMemcpyHostToDevice); cudacheck(err);
			err = hipMemcpy(d_b, b, sizeof_const, hipMemcpyHostToDevice); cudacheck(err);
			err = hipMemcpy(d_hash_table, hash_table, sizeof_hashtable, hipMemcpyHostToDevice); cudacheck(err);
			err = hipMemcpy(d_evict, evict, sizeof_input, hipMemcpyHostToDevice); cudacheck(err);
			err = hipMemcpy(d_hash_fun_i, hash_fun_i, sizeof_input, hipMemcpyHostToDevice); cudacheck(err);
			err = hipMemcpy(d_hash_fun, hash_fun, sizeof_input*t, hipMemcpyHostToDevice); cudacheck(err);
		}
		
		if (needhash == true)
		{
			hipEventRecord(start);
			hashcal << <set_size / 256 + 1, 256 >> > (d_input, d_hash_fun, d_a, d_b, table_size, set_size, t);
			hipEventRecord(stop);
			hipEventSynchronize(stop);
			hipEventElapsedTime(&elapsedTime, start, stop);
			stk.push(elapsedTime);
			//printf("hashcal using time��%f <ms>\n", elapsedTime);
		}
		

		hipEventRecord(start);
		hashfun << <set_size/256+1, 256 >> >(d_input, d_hash_table,d_hash_fun_i, d_a, d_b, table_size, d_evict,set_size, d_hash_fun, t);
		hipEventRecord(stop);

		hipEventSynchronize(stop);
		elapsedTime = 0;
		hipEventElapsedTime(&elapsedTime, start, stop);
		stk.push(elapsedTime);
		//printf("hashfun using time��%f <ms>\n", elapsedTime);
		 
		hipEventRecord(start);
		evicter << <set_size / 256+1, 256 >> >(d_hash_table,  d_hash_fun_i,d_input, d_evict, d_a, d_b, table_size, t, set_size,d_outnum, d_hash_fun);
		hipEventRecord(stop);

		//needhash = false;

		hipEventSynchronize(stop);
		elapsedTime = 0;
		hipEventElapsedTime(&elapsedTime, start, stop);
		stk.push(elapsedTime);
		//printf("evicter using time��%f <ms>\n", elapsedTime);

		err = hipMemcpy(&outnum, d_outnum, sizeof(int), hipMemcpyDeviceToHost); cudacheck(err);

		cout << "round is "<< round <<" out is "<<outnum<<endl;
		round++;
		
	}


	hipEventDestroy(start);
	hipEventDestroy(stop);
	float tim = 0;
	while (!stk.empty())
	{
		tim += stk.top();
		stk.pop();
	}
	cout << "time is " << tim<<" ms"<<endl;
	err = hipMemcpy(hash_table, d_hash_table, sizeof_hashtable, hipMemcpyDeviceToHost); cudacheck(err);
	err = hipMemcpy(hash_fun_i, d_hash_fun_i, sizeof_input, hipMemcpyDeviceToHost); cudacheck(err);

	checkright(hash_table,hash_fun_i,input,set_size, table_size,a,b);

	while(!stk.empty())
    {
        cout<<stk.top()<<endl;
        stk.pop();
    }

	hipFree(d_outnum);
	hipFree(d_input);
	hipFree(d_hash_table);
	hipFree(d_hash_fun_i);
	hipFree(d_a); hipFree(d_b);
	hipFree(d_evict);
	hipFree(d_hash_fun);
	//delete[] input;
	delete[] hash_table;
	delete[] hash_fun_i; 
	delete[] evict;
	delete[] hash_fun;
	return 0;
}

//int main() {
//	for (int i = 1; i <= 5; i++)
//	{
//	std::default_random_engine random(time(NULL));
//	std::uniform_int_distribution<int> ranin(1, pow(2, 25));
//	int * input = new int[pow(2, 24)];
//	for (int i = 0; i < pow(2,24); i++)
//		input[i] = ranin(random);
//	
//	float k[1] = { 1.2 };
//	
//	for (int j = 0; j <=0; j += 1) {
//		cout << "table_size is " << j << "n" << endl;
//		int table_size = (int)(pow(2, 24) *k[j]);
//		int bound = (int)4*log(table_size);
//		cout << log(table_size) << endl;
//			qwq(table_size, 24, input,bound);
//		}
//		cout << "---------------------" << endl;
//	delete[] input;
//	}
//}

//int main() {
//	for (int i = 1; i <= 5; i++)
//	{
//		std::default_random_engine random(time(NULL));
//		std::uniform_int_distribution<int> ranin(1, pow(2, 25));
//		int * input = new int[pow(2, 24)];
//		for (int i = 0; i < pow(2, 24); i++)
//			input[i] = ranin(random);
//		float k[3] = { 1.01,1.02,1.05 };
//		for (int j = 0; j <= 2; j += 1) {
//			cout << "table_size is " << j << "n" << endl;
//			int table_size = (int)(pow(2, 24) *k[j]);
//			qwq(table_size, 24, input);
//		}
//		cout << "---------------------" << endl;
//		delete[] input;
//	}
//}

//int main() {
//for (int i = 1; i <= 5; i++)
//	{
//	for (int j = 10; j <= 24; j++) {
//	cout << "the set size is " << j<<endl;
//	qwq(25, j);
//	}
//	cout << "---------------------" << endl;
//	}
//}

//int main() {
//for (int i = 1; i <= 5; i++)
//		{
//		std::uniform_int_distribution<int> ranin(1, pow(2, 25));
//		std::uniform_int_distribution<int> ranchoose(0, pow(2, 24));
//		std::default_random_engine random(time(NULL));
//		int set_size = pow(2, 24);
//		//cout << "the set size is " << j<<endl;
//		
//			int * inputold = new int[set_size];
//			
//			for (int k = 0; i < set_size; i++)
//				inputold[i] = ranin(random);
//			for (int j = 100; j >= 0; j -= 10)
//			{
//				int * input = new int[set_size];
//				cout << set_size * j / 100 << endl;
//				for (int k = 0; k < set_size*j/100; k++)
//					input[k] = inputold[ranchoose(random)];
//				for (int k = set_size * j / 100 ; k < set_size; k++)
//					input[k] = ranin(random);
//
//				qwq(25, 24, input);
//				delete[] input;
//			}
//			
//			delete[] inputold;
//			
//
//		}
//		cout << "---------------------" << endl;
//		
//
//	
//
//}